#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void camera_projection_kernel(
    const float* xyz,
    const float* K,
    const int N,
    float* uv
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    // u = fx * X / Z + cx
    uv[i * 2 + 0] = K[0] * xyz[i * 3 + 0] / xyz[i * 3 + 2] + K[2];
    // v = fy * Y / Z + cy
    uv[i * 2 + 1] = K[4] * xyz[i * 3 + 1] / xyz[i * 3 + 2] + K[5];
}

void camera_projection_cuda(
    torch::Tensor xyz,
    torch::Tensor K,
    torch::Tensor uv
) {
    TORCH_CHECK(xyz.is_cuda(), "xyz must be a CUDA tensor");
    TORCH_CHECK(K.is_cuda(), "K must be a CUDA tensor");
    TORCH_CHECK(uv.is_cuda(), "uv must be a CUDA tensor");

    TORCH_CHECK(xyz.is_contiguous(), "xyz must be contiguous");
    TORCH_CHECK(K.is_contiguous(), "K must be contiguous");
    TORCH_CHECK(uv.is_contiguous(), "uv must be contiguous");

    const int N = xyz.size(0);
    TORCH_CHECK(xyz.size(1) == 3, "xyz must have shape Nx3");
    TORCH_CHECK(K.size(0) == 3, "K must have shape 3x3");
    TORCH_CHECK(K.size(1) == 3, "K must have shape 3x3");
    TORCH_CHECK(uv.size(0) == N, "uv must have shape Nx2");
    TORCH_CHECK(uv.size(1) == 2, "uv must have shape Nx2");

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    camera_projection_kernel<<<gridsize, blocksize>>>(
        xyz.data_ptr<float>(),
        K.data_ptr<float>(),
        N,
        uv.data_ptr<float>()
    );
    hipDeviceSynchronize();
}


__global__ void compute_sigma_world_kernel(
    const float* quaternions,
    const float* scales,
    const int N,
    float* sigma_world
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    float qw = quaternions[i * 4 + 0];
    float qx = quaternions[i * 4 + 1];
    float qy = quaternions[i * 4 + 2];
    float qz = quaternions[i * 4 + 3];

    float norm = sqrt(qx * qx + qy * qy + qz * qz + qw * qw);

    // zero magnitude quaternion is not valid
    // (TODO) - how to handle this?
    assert(abs(norm) > 1e-6); 
    qx /= norm;
    qy /= norm;
    qz /= norm;
    qw /= norm;

    float r00 = 1 - 2 * qy * qy - 2 * qz * qz;  
    float r01 = 2 * qx * qy - 2 * qz * qw;
    float r02 = 2 * qx * qz + 2 * qy * qw;
    float r10 = 2 * qx * qy + 2 * qz * qw;
    float r11 = 1 - 2 * qx * qx - 2 * qz * qz;
    float r12 = 2 * qy * qz - 2 * qx * qw;
    float r20 = 2 * qx * qz - 2 * qy * qw;
    float r21 = 2 * qy * qz + 2 * qx * qw;
    float r22 = 1 - 2 * qx * qx - 2 * qy * qy;

    float sx = __expf(scales[i * 3 + 0]);
    float sy = __expf(scales[i * 3 + 1]);
    float sz = __expf(scales[i * 3 + 2]);

    float sx_sq = sx * sx;
    float sy_sq = sy * sy;
    float sz_sq = sz * sz;

    sigma_world[i * 9 + 0] = r00*r00*sx_sq + r01*r01*sy_sq + r02*r02*sz_sq;
    sigma_world[i * 9 + 1] = r00*r10*sx_sq + r01*r11*sy_sq + r02*r12*sz_sq;
    sigma_world[i * 9 + 2] = r00*r20*sx_sq + r01*r21*sy_sq + r02*r22*sz_sq;

    sigma_world[i * 9 + 3] = r00*r10*sx_sq + r01*r11*sy_sq + r02*r12*sz_sq;
    sigma_world[i * 9 + 4] = r10*r10*sx_sq + r11*r11*sy_sq + r12*r12*sz_sq;
    sigma_world[i * 9 + 5] = r10*r20*sx_sq + r11*r21*sy_sq + r12*r22*sz_sq;

    sigma_world[i * 9 + 6] = r00*r20*sx_sq + r01*r21*sy_sq + r02*r22*sz_sq;
    sigma_world[i * 9 + 7] = r10*r20*sx_sq + r11*r21*sy_sq + r12*r22*sz_sq;
    sigma_world[i * 9 + 8] = r20*r20*sx_sq + r21*r21*sy_sq + r22*r22*sz_sq;
}


void compute_sigma_world_cuda(
    torch::Tensor quaternions,
    torch::Tensor scales,
    torch::Tensor sigma_world
) {
    TORCH_CHECK(quaternions.is_cuda(), "quaternions must be a CUDA tensor");
    TORCH_CHECK(scales.is_cuda(), "scales must be a CUDA tensor");
    TORCH_CHECK(sigma_world.is_cuda(), "sigma_world must be a CUDA tensor");

    TORCH_CHECK(quaternions.is_contiguous(), "quaternions must be contiguous");
    TORCH_CHECK(scales.is_contiguous(), "scales must be contiguous");
    TORCH_CHECK(sigma_world.is_contiguous(), "sigma_world must be contiguous");

    const int N = quaternions.size(0);
    TORCH_CHECK(quaternions.size(1) == 4, "quaternions must have shape Nx4");
    TORCH_CHECK(scales.size(0) == N, "scales must have shape Nx1");
    TORCH_CHECK(sigma_world.size(0) == N, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(sigma_world.size(1) == 3, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(sigma_world.size(2) == 3, "sigma_world must have shape Nx3x3");


    // can probably update this to improve perf
    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    compute_sigma_world_kernel<<<gridsize, blocksize>>>(
        quaternions.data_ptr<float>(),
        scales.data_ptr<float>(),
        N,
        sigma_world.data_ptr<float>()
    );
    hipDeviceSynchronize();
}


__global__ void compute_projection_jacobian_kernel(
    const float* xyz,
    const float fx,
    const float fy,
    const int N,
    float* J
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    float x = xyz[i * 3 + 0];
    float y = xyz[i * 3 + 1];
    float z = xyz[i * 3 + 2];

    float z_inv = 1.0 / z;

    J[i * 6 + 0] = fx * z_inv;
    J[i * 6 + 1] = 0;
    J[i * 6 + 2] = -fx * x * z_inv * z_inv;
    J[i * 6 + 3] = 0;
    J[i * 6 + 4] = fy * z_inv;
    J[i * 6 + 5] = -fy * y * z_inv * z_inv;
}


void compute_projection_jacobian_cuda(
    torch::Tensor xyz,
    const float fx,
    const float fy,
    torch::Tensor J
) {
    TORCH_CHECK(xyz.is_cuda(), "xyz must be a CUDA tensor");
    TORCH_CHECK(J.is_cuda(), "J must be a CUDA tensor");

    TORCH_CHECK(xyz.is_contiguous(), "xyz must be contiguous");
    TORCH_CHECK(J.is_contiguous(), "J must be contiguous");

    const int N = xyz.size(0);
    TORCH_CHECK(xyz.size(1) == 3, "xyz must have shape Nx3");
    TORCH_CHECK(J.size(0) == N, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(1) == 2, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(2) == 3, "J must have shape Nx2x3");

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    compute_projection_jacobian_kernel<<<gridsize, blocksize>>>(
        xyz.data_ptr<float>(),
        fx,
        fy,
        N,
        J.data_ptr<float>()
    );
}

// templated matrix multiplication function for row major matrices
template <typename T>
__device__ void matrix_multiply(
    const T* A,
    const T* B,
    T* C,
    int num_rows_A,
    int num_cols_A,
    int num_cols_B) {
    
    #pragma unroll
    for (int row_a = 0; row_a < num_rows_A; row_a++) {
        #pragma unroll
        for (int col_b = 0; col_b < num_cols_B; col_b++) {
            T sum = 0;
            #pragma unroll
            for (int cols_A = 0; cols_A < num_cols_A; cols_A++) {
                sum += A[row_a * num_cols_A + cols_A] * B[cols_A * num_cols_B + col_b];
            }
            C[row_a * num_cols_B + col_b] = sum;
        }
    }
}


__global__ void compute_sigma_image_kernel(
    const float* sigma_world,
    const float* J,
    const float* world_T_image,
    const int N,
    float* sigma_image
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    // get rotation matrix
    float W[9];
    W[0] = world_T_image[0];
    W[1] = world_T_image[1];
    W[2] = world_T_image[2];
    W[3] = world_T_image[4];
    W[4] = world_T_image[5];
    W[5] = world_T_image[6];
    W[6] = world_T_image[8];
    W[7] = world_T_image[9];
    W[8] = world_T_image[10];

    // compute JW = J * W)
    float JW[6];
    matrix_multiply<float>(J + i * 6, W, JW, 2, 3, 3);

    // compute JWSigma = JW * sigma_world
    float JWSigma[6];
    matrix_multiply<float>(JW, sigma_world + i * 9, JWSigma, 2, 3, 3);

    float JW_T[6];
    JW_T[0] = JW[0];
    JW_T[1] = JW[3];
    JW_T[2] = JW[1];
    JW_T[3] = JW[4];
    JW_T[4] = JW[2];
    JW_T[5] = JW[5];

    // compute sigma_image = JWSigma * JW^T
    // (TODO) transpose JW -> JW_T inplace for better perf?
    matrix_multiply<float>(JWSigma, JW_T, sigma_image + i * 4, 2, 3, 2);
}


void compute_sigma_image_cuda(
    torch::Tensor sigma_world,
    torch::Tensor J,
    torch::Tensor world_T_image,
    torch::Tensor sigma_image
) {
    TORCH_CHECK(sigma_world.is_cuda(), "sigma_world must be a CUDA tensor");
    TORCH_CHECK(J.is_cuda(), "J must be a CUDA tensor");
    TORCH_CHECK(world_T_image.is_cuda(), "world_T_image must be a CUDA tensor");
    TORCH_CHECK(sigma_image.is_cuda(), "sigma_image must be a CUDA tensor");

    TORCH_CHECK(sigma_world.is_contiguous(), "sigma_world must be contiguous");
    TORCH_CHECK(J.is_contiguous(), "J must be contiguous");
    TORCH_CHECK(world_T_image.is_contiguous(), "world_T_image must be contiguous");
    TORCH_CHECK(sigma_image.is_contiguous(), "sigma_image must be contiguous");

    const int N = sigma_world.size(0);
    TORCH_CHECK(sigma_world.size(1) == 3, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(sigma_world.size(2) == 3, "sigma_world must have shape Nx3x3");
    TORCH_CHECK(J.size(0) == N, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(1) == 2, "J must have shape Nx2x3");
    TORCH_CHECK(J.size(2) == 3, "J must have shape Nx2x3");
    TORCH_CHECK(world_T_image.size(0) == 4, "world_T_image must have shape 4x4");
    TORCH_CHECK(world_T_image.size(1) == 4, "world_T_image must have shape 4x4");
    TORCH_CHECK(sigma_image.size(0) == N, "sigma_image must have shape Nx2x2");
    TORCH_CHECK(sigma_image.size(1) == 2, "sigma_image must have shape Nx2x2");
    TORCH_CHECK(sigma_image.size(2) == 2, "sigma_image must have shape Nx2x2");

    const int max_threads_per_block = 1024;
    const int num_blocks = (N + max_threads_per_block - 1) / max_threads_per_block;
    dim3 gridsize(num_blocks, 1, 1);
    dim3 blocksize(max_threads_per_block, 1, 1);

    compute_sigma_image_kernel<<<gridsize, blocksize>>>(
        sigma_world.data_ptr<float>(),
        J.data_ptr<float>(),
        world_T_image.data_ptr<float>(),
        N,
        sigma_image.data_ptr<float>()
    );
}
