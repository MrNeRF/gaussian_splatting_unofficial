#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template<typename T>
__device__ T compute_norm_prob(
    const int gaussian_idx,
    const int u_splat,
    const int v_splat,
    const T* __restrict__ uvs,
    const T* __restrict__ sigma_image,
    bool use_fast_exp
) {
    const T u_mean = uvs[gaussian_idx * 2 + 0];
    const T v_mean = uvs[gaussian_idx * 2 + 1];

    const T u_diff = T(u_splat) - u_mean;
    const T v_diff = T(v_splat) - v_mean;

    // 2d covariance matrix
    const T a = sigma_image[gaussian_idx * 4 + 0];
    const T b = sigma_image[gaussian_idx * 4 + 1];
    const T c = sigma_image[gaussian_idx * 4 + 2];
    const T d = sigma_image[gaussian_idx * 4 + 3];

    T det = a * d - b * c;
    if (det < 0.0) {
        return 0.0;
    }
    if (det < 1e-14) {
        det += 1e-14;
    }
    // compute mahalanobis distance
    const T mh_sq = (d * u_diff * u_diff - (b + c) * u_diff * v_diff + a * v_diff * v_diff) / det;
    if (mh_sq < 0.0) {
        return 0.0;
    }
    // probablity at this pixel normalized to have probability at the center of the gaussian to be 1.0
    T norm_prob = 0.0;
    if (use_fast_exp) {
        norm_prob = __expf(-0.5 * mh_sq);
    } else {
        norm_prob = exp(-0.5 * mh_sq);
    }
    return norm_prob;
}


template<typename T>
__global__ void render_tiles_backward_kernel(
        const T* __restrict__ uvs,
        const T* __restrict__ opacity,
        const T* __restrict__ rgb,
        const T* __restrict__ sigma_image,
        const int* __restrict__ splat_start_end_idx_by_tile_idx,
        const int* __restrict__ gaussian_idx_by_splat_idx,
        const int* __restrict__ num_splats_per_pixel,
        const T* __restrict__ final_weight_per_pixel,
        const T* __restrict__ grad_image,
        const int image_width,
        const int image_height,
        bool use_fast_exp,
        T* __restrict__ grad_rgb, // N_gaussians x 3
        T* __restrict__ grad_opacity, // N_gaussians x 1
        T* __restrict__ grad_uv, // N_gaussians x 2
        T* __restrict__ grad_sigma_image // N_gaussians x 4
) {
    // grid = tiles, blocks = pixels within each tile
    const int u_splat = blockIdx.x * blockDim.x + threadIdx.x;
    const int v_splat = blockIdx.y * blockDim.y + threadIdx.y;
    if (u_splat >= image_width || v_splat >= image_height) {
        return;
    }

    const int tile_idx = blockIdx.x + blockIdx.y * gridDim.x;
    const int splat_idx_start = splat_start_end_idx_by_tile_idx[tile_idx];
    const int splat_idx_end = splat_start_end_idx_by_tile_idx[tile_idx + 1];
    int num_splats = num_splats_per_pixel[v_splat * image_width + u_splat];
    if (num_splats == 0) {
        return;
    }

    T grad_image_r = grad_image[(v_splat * image_width + u_splat) * 3 + 0];
    T grad_image_g = grad_image[(v_splat * image_width + u_splat) * 3 + 1];
    T grad_image_b = grad_image[(v_splat * image_width + u_splat) * 3 + 2];

    T color_accum[3] = {0.0, 0.0, 0.0};
    T weight = final_weight_per_pixel[u_splat + v_splat * image_width];

    if (weight < 1e-14) {
        return;
    }
    for (int i = num_splats - 1; i >= 0; i--) {
        const int splat_idx = splat_idx_start + i;
        const int gaussian_idx = gaussian_idx_by_splat_idx[splat_idx];

        const T u_mean = uvs[gaussian_idx * 2 + 0];
        const T v_mean = uvs[gaussian_idx * 2 + 1];

        const T u_diff = T(u_splat) - u_mean;
        const T v_diff = T(v_splat) - v_mean;

        // 2d covariance matrix
        const T a = sigma_image[gaussian_idx * 4 + 0];
        const T b = sigma_image[gaussian_idx * 4 + 1];
        const T c = sigma_image[gaussian_idx * 4 + 2];
        const T d = sigma_image[gaussian_idx * 4 + 3];
        T det = a * d - b * c;

        T norm_prob = 0.0;
        if (det > 0.0) {
            if (det < 1e-14) {
                det += 1e-14;
            }
            // compute mahalanobis distance
            const T mh_sq = (d * u_diff * u_diff - (b + c) * u_diff * v_diff + a * v_diff * v_diff) / det;
            if (mh_sq > 0.0) {
                // probablity at this pixel normalized to have probability at the center of the gaussian to be 1.0
                if (use_fast_exp) {
                    norm_prob = __expf(-0.5 * mh_sq);
                } else {
                    norm_prob = exp(-0.5 * mh_sq);
                }
            }
        }
        
        T alpha = opacity[gaussian_idx] * norm_prob;

        // update weight
        if (i < num_splats - 1) {
            weight = weight / (1.0 - alpha);
        }

        // update rgb gradient. Since each gaussian is splat to multiple pixels, we need to use atomicAdd
        atomicAdd(grad_rgb + gaussian_idx * 3 + 0, alpha * weight * grad_image_r);
        atomicAdd(grad_rgb + gaussian_idx * 3 + 1, alpha * weight * grad_image_g);
        atomicAdd(grad_rgb + gaussian_idx * 3 + 2, alpha * weight * grad_image_b);
        
        // prevent divide by zero
        if (abs(alpha - 1.0) < 1e-14) {
            alpha -= 1e-14;
        }
        T grad_alpha_r = (rgb[gaussian_idx * 3 + 0] * weight - color_accum[0]/(1.0 - alpha)) * grad_image_r;
        T grad_alpha_g = (rgb[gaussian_idx * 3 + 1] * weight - color_accum[1]/(1.0 - alpha)) * grad_image_g;
        T grad_alpha_b = (rgb[gaussian_idx * 3 + 2] * weight - color_accum[2]/(1.0 - alpha)) * grad_image_b;
        T grad_alpha = grad_alpha_r + grad_alpha_g + grad_alpha_b;

        T grad_opa = norm_prob * grad_alpha;

        // update opacity gradient
        atomicAdd(grad_opacity + gaussian_idx, grad_opa);

        // compute gradient for probability
        T grad_prob = opacity[gaussian_idx] * grad_alpha;
        T grad_mh_sq = -0.5 * norm_prob * grad_prob;

        // compute gradient for projected mean
        T grad_u = -(-b * v_diff - c * v_diff + 2 * d * u_diff) / det * grad_mh_sq;
        T grad_v = -(2 * a * v_diff - b * u_diff - c * u_diff) / det * grad_mh_sq;
        atomicAdd(grad_uv + gaussian_idx * 2 + 0, grad_u);
        atomicAdd(grad_uv + gaussian_idx * 2 + 1, grad_v);

        T grad_a = (-d * (a * v_diff * v_diff - b * u_diff * v_diff - c * u_diff * v_diff + d * u_diff * u_diff) / (det * det)  + (v_diff * v_diff) / det) * grad_mh_sq;
        T grad_b = (c * (a * v_diff * v_diff - b * u_diff * v_diff - c * u_diff * v_diff + d * u_diff * u_diff) / (det * det)  - (u_diff * v_diff) / det) * grad_mh_sq;
        T grad_c = (b * (a * v_diff * v_diff - b * u_diff * v_diff - c * u_diff * v_diff + d * u_diff * u_diff) / (det * det)  - (u_diff * v_diff) / det) * grad_mh_sq;
        T grad_d = (-a * (a * v_diff * v_diff - b * u_diff * v_diff - c * u_diff * v_diff + d * u_diff * u_diff) / (det * det)  + (u_diff * u_diff) / det) * grad_mh_sq;
        atomicAdd(grad_sigma_image + gaussian_idx * 4 + 0, grad_a);
        atomicAdd(grad_sigma_image + gaussian_idx * 4 + 1, grad_b);
        atomicAdd(grad_sigma_image + gaussian_idx * 4 + 2, grad_c);
        atomicAdd(grad_sigma_image + gaussian_idx * 4 + 3, grad_d);

        // update color_accum for next splat
        T update = rgb[gaussian_idx * 3 + 0] * alpha * weight;
        color_accum[0] += update;
        color_accum[1] += rgb[gaussian_idx * 3 + 1] * alpha * weight;
        color_accum[2] += rgb[gaussian_idx * 3 + 2] * alpha * weight;
    }
}


void render_tiles_backward_cuda(
    torch::Tensor uvs,
    torch::Tensor opacity,
    torch::Tensor rgb,
    torch::Tensor sigma_image,
    torch::Tensor splat_start_end_idx_by_tile_idx,
    torch::Tensor gaussian_idx_by_splat_idx,
    torch::Tensor num_splats_per_pixel,
    torch::Tensor final_weight_per_pixel,
    torch::Tensor grad_image,
    torch::Tensor grad_rgb,
    torch::Tensor grad_opacity,
    torch::Tensor grad_uv,
    torch::Tensor grad_sigma_image
) {
    TORCH_CHECK(uvs.device().is_cuda(), "uvs must be a CUDA tensor");
    TORCH_CHECK(opacity.device().is_cuda(), "opacity must be a CUDA tensor");
    TORCH_CHECK(rgb.device().is_cuda(), "rgb must be a CUDA tensor");
    TORCH_CHECK(sigma_image.device().is_cuda(), "sigma_image must be a CUDA tensor");
    TORCH_CHECK(splat_start_end_idx_by_tile_idx.device().is_cuda(), "splat_start_end_idx_by_tile_idx must be a CUDA tensor");
    TORCH_CHECK(gaussian_idx_by_splat_idx.device().is_cuda(), "gaussian_idx_by_splat_idx must be a CUDA tensor");
    TORCH_CHECK(num_splats_per_pixel.device().is_cuda(), "num_splats_per_pixel must be a CUDA tensor");
    TORCH_CHECK(final_weight_per_pixel.device().is_cuda(), "final_weight_per_pixel must be a CUDA tensor");
    TORCH_CHECK(grad_image.device().is_cuda(), "grad_image must be a CUDA tensor");
    TORCH_CHECK(grad_rgb.device().is_cuda(), "grad_rgb must be a CUDA tensor");
    TORCH_CHECK(grad_opacity.device().is_cuda(), "grad_opacity must be a CUDA tensor");
    TORCH_CHECK(grad_uv.device().is_cuda(), "grad_uv must be a CUDA tensor");
    TORCH_CHECK(grad_sigma_image.device().is_cuda(), "grad_sigma_image must be a CUDA tensor");

    TORCH_CHECK(uvs.is_contiguous(), "uvs must be contiguous");
    TORCH_CHECK(opacity.is_contiguous(), "opacity must be contiguous");
    TORCH_CHECK(rgb.is_contiguous(), "rgb must be contiguous");
    TORCH_CHECK(sigma_image.is_contiguous(), "sigma_image must be contiguous");
    TORCH_CHECK(splat_start_end_idx_by_tile_idx.is_contiguous(), "splat_start_end_idx_by_tile_idx must be contiguous");
    TORCH_CHECK(gaussian_idx_by_splat_idx.is_contiguous(), "gaussian_idx_by_splat_idx must be contiguous");
    TORCH_CHECK(num_splats_per_pixel.is_contiguous(), "num_splats_per_pixel must be contiguous");
    TORCH_CHECK(final_weight_per_pixel.is_contiguous(), "final_weight_per_pixel must be contiguous");
    TORCH_CHECK(grad_image.is_contiguous(), "grad_image must be contiguous");
    TORCH_CHECK(grad_rgb.is_contiguous(), "grad_rgb must be contiguous");
    TORCH_CHECK(grad_opacity.is_contiguous(), "grad_opacity must be contiguous");
    TORCH_CHECK(grad_uv.is_contiguous(), "grad_uv must be contiguous");
    TORCH_CHECK(grad_sigma_image.is_contiguous(), "grad_sigma_image must be contiguous");

    int N = uvs.size(0);
    TORCH_CHECK(uvs.size(1) == 2, "uvs must have 2 channels");
    TORCH_CHECK(opacity.size(0) == N, "opacity must have the same size as uvs");
    TORCH_CHECK(rgb.size(0) == N, "rgb must have the same size as uvs");
    TORCH_CHECK(rgb.size(1) == 3, "rgb must have 3 channels");
    TORCH_CHECK(sigma_image.size(0) == N, "sigma_image must have the same size as uvs");
    TORCH_CHECK(sigma_image.size(1) == 2, "sigma_image must have 2x2 channels");
    TORCH_CHECK(sigma_image.size(2) == 2, "sigma_image must have 2x2 channels");
    

    int image_height = num_splats_per_pixel.size(0);
    int image_width = num_splats_per_pixel.size(1);

    int num_tiles_x = (image_width + 16 - 1) / 16;
    int num_tiles_y = (image_height + 16 - 1) / 16;

    TORCH_CHECK(splat_start_end_idx_by_tile_idx.size(0) == num_tiles_x * num_tiles_y + 1, "splat_start_end_idx_by_tile_idx ");
    TORCH_CHECK(num_splats_per_pixel.size(0) == image_height, "num_splats_per_pixel must have the same size as the image");
    TORCH_CHECK(num_splats_per_pixel.size(1) == image_width, "num_splats_per_pixel must have the same size as the image");
    TORCH_CHECK(final_weight_per_pixel.size(0) == image_height, "final_weight_per_pixel must have the same size as the image");
    TORCH_CHECK(final_weight_per_pixel.size(1) == image_width, "final_weight_per_pixel must have the same size as the image");
    TORCH_CHECK(grad_image.size(0) == image_height, "grad_image must have the same size as the image");
    TORCH_CHECK(grad_image.size(1) == image_width, "grad_image must have the same size as the image");
    TORCH_CHECK(grad_image.size(2) == 3, "grad_image must have 3 channels");

    dim3 block_size(16, 16, 1);
    dim3 grid_size(num_tiles_x, num_tiles_y, 1);

    if (uvs.dtype() == torch::kFloat32) {
        render_tiles_backward_kernel<float><<<grid_size, block_size>>>(
            uvs.data_ptr<float>(),
            opacity.data_ptr<float>(),
            rgb.data_ptr<float>(),
            sigma_image.data_ptr<float>(),
            splat_start_end_idx_by_tile_idx.data_ptr<int>(),
            gaussian_idx_by_splat_idx.data_ptr<int>(),
            num_splats_per_pixel.data_ptr<int>(),
            final_weight_per_pixel.data_ptr<float>(),
            grad_image.data_ptr<float>(),
            image_width,
            image_height,
            true,
            grad_rgb.data_ptr<float>(),
            grad_opacity.data_ptr<float>(),
            grad_uv.data_ptr<float>(),
            grad_sigma_image.data_ptr<float>()
        );
    } else if (uvs.dtype() == torch::kFloat64) {
        render_tiles_backward_kernel<double><<<grid_size, block_size>>>(
            uvs.data_ptr<double>(),
            opacity.data_ptr<double>(),
            rgb.data_ptr<double>(),
            sigma_image.data_ptr<double>(),
            splat_start_end_idx_by_tile_idx.data_ptr<int>(),
            gaussian_idx_by_splat_idx.data_ptr<int>(),
            num_splats_per_pixel.data_ptr<int>(),
            final_weight_per_pixel.data_ptr<double>(),
            grad_image.data_ptr<double>(),
            image_width,
            image_height,
            false,
            grad_rgb.data_ptr<double>(),
            grad_opacity.data_ptr<double>(),
            grad_uv.data_ptr<double>(),
            grad_sigma_image.data_ptr<double>()
        );
    } else {
        AT_ERROR("Unsupported dtype");
    }
    hipDeviceSynchronize();
}